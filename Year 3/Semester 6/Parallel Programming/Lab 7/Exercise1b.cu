
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 4

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // Initialize input vectors a and b
    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));

    // Copy data from host to device
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // Define block size and grid size
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // Perform vector addition on GPU
    vectorAdd<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, N);

    // Copy result back to host
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // Display the result
    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}
