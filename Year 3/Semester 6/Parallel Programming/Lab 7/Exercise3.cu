
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MASK_WIDTH 3
#define TILE_WIDTH 32

__global__ void convolutionKernel(float *N, float *M, float *P, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int radius = MASK_WIDTH / 2;
    float result = 0.0f;

    for (int i = -radius; i <= radius; i++) {
        int index = idx + i;
        if (index >= 0 && index < width) {
            result += N[index] * M[i + radius];
        }
    }

    P[idx] = result;
}

int main() {
    int width = 10; // size of input array
    int mask_width = MASK_WIDTH;
    int size = width * sizeof(float);
    float *h_N, *h_M, *h_P; // host arrays
    float *d_N, *d_M, *d_P; // device arrays

    // Allocate memory on the host
    h_N = (float*)malloc(size);
    h_M = (float*)malloc(mask_width * sizeof(float));
    h_P = (float*)malloc(size);

    // Initialize input array and mask array
    for (int i = 0; i < width; i++) {
        h_N[i] = i; // some example data
    }
    for (int i = 0; i < mask_width; i++) {
        h_M[i] = 1.0f; // simple mask for averaging
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_N, size);
    hipMalloc((void**)&d_M, mask_width * sizeof(float));
    hipMalloc((void**)&d_P, size);

    // Copy data from host to device
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, mask_width * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimGrid((width + TILE_WIDTH - 1) / TILE_WIDTH, 1, 1);
    dim3 dimBlock(TILE_WIDTH, 1, 1);

    // Launch the CUDA kernel
    convolutionKernel<<<dimGrid, dimBlock>>>(d_N, d_M, d_P, width);

    // Copy result back to host
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);

    // Print the result array
    printf("Result of convolution:\n");
    for (int i = 0; i < width; i++) {
        printf("%.2f ", h_P[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);

    // Free host memory
    free(h_N);
    free(h_M);
    free(h_P);

    return 0;
}
