
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 4
#define THREADS_PER_BLOCK 256

__global__ void computeSine(float *angles, float *sineValues, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        sineValues[index] = sinf(angles[index]);
    }
}

int main() {
    float angles[N], sineValues[N];
    float *dev_angles, *dev_sineValues;

    // Initialize input array with angles in radians
    for (int i = 0; i < N; ++i) {
        angles[i] = i * (3.14159 / N);  // Converting to radians from 0 to pi
    }

    // Allocate device memory
    hipMalloc((void **)&dev_angles, N * sizeof(float));
    hipMalloc((void **)&dev_sineValues, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(dev_angles, angles, N * sizeof(float), hipMemcpyHostToDevice);

    // Define block size and grid size
    int blockSize = THREADS_PER_BLOCK;
    int gridSize = (N + blockSize - 1) / blockSize;

    // Perform computation on GPU
    computeSine<<<gridSize, blockSize>>>(dev_angles, dev_sineValues, N);

    // Copy result back to host
    hipMemcpy(sineValues, dev_sineValues, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(dev_angles);
    hipFree(dev_sineValues);

    // Display the result
    printf("Angle (radians)\tSine Value\n");
    for (int i = 0; i < N; ++i) {
        printf("%f\t%f\n", angles[i], sineValues[i]);
    }

    return 0;
}
