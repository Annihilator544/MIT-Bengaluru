
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10
#define BLOCK_SIZE 256

__global__ void oddEvenSort(int *arr, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        int temp;
        if (i % 2 == 0) { // Even phase
            if (i < n - 1 && arr[i] > arr[i + 1]) {
                temp = arr[i];
                arr[i] = arr[i + 1];
                arr[i + 1] = temp;
            }
        } else { // Odd phase
            if (i > 0 && arr[i] < arr[i - 1]) {
                temp = arr[i];
                arr[i] = arr[i - 1];
                arr[i - 1] = temp;
            }
        }
    }
}

int main() {
    int arr[N];

    // Initialize array with random values
    for (int i = 0; i < N; i++) {
        arr[i] = rand() % 10; // Random values between 0 and 999
    }

    int *d_arr;
    hipMalloc((void**)&d_arr, N * sizeof(int));
    hipMemcpy(d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 blocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1);
    dim3 threads(BLOCK_SIZE, 1, 1);

    // Perform odd-even sort in parallel
    for (int i = 0; i < N; i++) {
        oddEvenSort<<<blocks, threads>>>(d_arr, N);
        hipDeviceSynchronize();
    }

    hipMemcpy(arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);

    // Print sorted array (for verification)
    printf("Sorted Array:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");

    return 0;
}
