
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N_ROWS 4
#define N_COLS 5

__global__ void selectionSortRows(float *matrix, int numCols) {
    int rowIdx = blockIdx.x;
    int startIdx = rowIdx * numCols;

    for (int i = 0; i < numCols - 1; i++) {
        int minIdx = i;
        for (int j = i + 1; j < numCols; j++) {
            if (matrix[startIdx + j] < matrix[startIdx + minIdx]) {
                minIdx = j;
            }
        }
        // Swap elements
        float temp = matrix[startIdx + i];
        matrix[startIdx + i] = matrix[startIdx + minIdx];
        matrix[startIdx + minIdx] = temp;
    }
}

int main() {
    float h_matrix[N_ROWS][N_COLS] = {
        {5, 3, 1, 4, 2},
        {9, 6, 8, 7, 10},
        {15, 13, 11, 14, 12},
        {19, 16, 18, 17, 20}
    };

    int size = N_ROWS * N_COLS * sizeof(float);
    float *d_matrix;

    // Allocate memory on the device
    hipMalloc((void**)&d_matrix, size);

    // Copy data from host to device
    hipMemcpy(d_matrix, h_matrix, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 gridDim(N_ROWS, 1, 1);

    // Launch the CUDA kernel to sort each row
    selectionSortRows<<<gridDim, 1>>>(d_matrix, N_COLS);

    // Copy the sorted data back to host
    hipMemcpy(h_matrix, d_matrix, size, hipMemcpyDeviceToHost);

    // Print the sorted matrix (for verification)
    printf("Sorted Matrix:\n");
    for (int i = 0; i < N_ROWS; i++) {
        for (int j = 0; j < N_COLS; j++) {
            printf("%.2f ", h_matrix[i][j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_matrix);

    return 0;
}
