
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(float *x, float *y, float alpha, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        y[idx] = alpha * x[idx] + y[idx];
    }
}

int main() {
    int n = 10; // size of vectors
    float alpha = 2.0f; // scalar value
    int size = n * sizeof(float);
    float *h_x, *h_y; // host arrays
    float *d_x, *d_y; // device arrays

    // Allocate memory on the host
    h_x = (float*)malloc(size);
    h_y = (float*)malloc(size);

    // Initialize input vectors x and y
    for (int i = 0; i < n; i++) {
        h_x[i] = i; // some example data
        h_y[i] = i * 2; // some example data
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);

    // Copy data from host to device
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Launch the CUDA kernel
    vectorAdd<<<numBlocks, blockSize>>>(d_x, d_y, alpha, n);

    // Copy result back to host
    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

    // Print the result array (for verification)
    printf("Result of y = αx + y:\n");
    for (int i = 0; i < n; i++) {
        printf("%.2f ", h_y[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);

    // Free host memory
    free(h_x);
    free(h_y);

    return 0;
}
