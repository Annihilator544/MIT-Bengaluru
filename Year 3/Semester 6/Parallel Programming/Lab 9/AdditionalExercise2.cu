
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 4
#define THREADS_PER_BLOCK 256

// CUDA kernel to calculate factorial
__device__ int factorial(int n) {
    if (n <= 1) return 1;
    return n * factorial(n - 1);
}

// CUDA kernel to calculate sum of digits
__device__ int sumOfDigits(int num) {
    int sum = 0;
    while (num != 0) {
        sum += num % 10;
        num /= 10;
    }
    return sum;
}

__global__ void processMatrix(int *matrix) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N * N) {
        int row = tid / N;
        int col = tid % N;
        int value = matrix[tid];

        if (row == col) {
            matrix[tid] = 0; // Replace principal diagonal elements with zero
        } else if (row < col) {
            // Calculate factorial for elements above the principal diagonal
            matrix[tid] = factorial(value);
        } else {
            // Calculate sum of digits for elements below the principal diagonal
            matrix[tid] = sumOfDigits(value);
        }
    }
}

int main() {
    int matrix[N][N] = {{1, 2, 3, 4},
                        {5, 6, 7, 8},
                        {9, 10, 11, 12},
                        {13, 14, 15, 16}};

    int *d_matrix;
    int size = N * N * sizeof(int);

    // Allocate memory for matrix A on device
    hipMalloc((void **)&d_matrix, size);

    // Copy matrix A to device
    hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice);

    int numBlocks = (N * N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel to process matrix elements
    processMatrix<<<numBlocks, THREADS_PER_BLOCK>>>(d_matrix);

    // Copy matrix back to host
    hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_matrix);

    // Display resultant matrix
    printf("Resultant Matrix:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", matrix[i][j]);
        }
        printf("\n");
    }

    return 0;
}
