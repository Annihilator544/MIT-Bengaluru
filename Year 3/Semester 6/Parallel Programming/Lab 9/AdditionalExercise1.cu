
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define M 3
#define N 3
#define THREADS_PER_BLOCK 256

__global__ void processMatrix(int *matrixA, int *matrixB, int rows, int cols) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < rows * cols) {
        int row = tid / cols;
        int col = tid % cols;
        int value = matrixA[tid];

        // Calculate row sum and column sum
        int rowSum = 0;
        int colSum = 0;
        for (int i = 0; i < cols; i++) {
            rowSum += matrixA[row * cols + i];
        }
        for (int i = 0; i < rows; i++) {
            colSum += matrixA[i * cols + col];
        }

        // Replace even elements with row sum, odd elements with column sum
        matrixB[tid] = (value % 2 == 0) ? rowSum : colSum;
    }
}

int main() {
    int matrixA[M][N] = {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}};
    int matrixB[M][N];

    int *d_matrixA, *d_matrixB;
    int size = M * N * sizeof(int);

    // Allocate memory for matrix A and matrix B on device
    hipMalloc((void **)&d_matrixA, size);
    hipMalloc((void **)&d_matrixB, size);

    // Copy matrix A to device
    hipMemcpy(d_matrixA, matrixA, size, hipMemcpyHostToDevice);

    int numBlocks = (M * N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel to process matrix elements
    processMatrix<<<numBlocks, THREADS_PER_BLOCK>>>(d_matrixA, d_matrixB, M, N);

    // Copy matrix B back to host
    hipMemcpy(matrixB, d_matrixB, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_matrixA);
    hipFree(d_matrixB);

    // Display resultant matrix B
    printf("Matrix B:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", matrixB[i][j]);
        }
        printf("\n");
    }

    return 0;
}
