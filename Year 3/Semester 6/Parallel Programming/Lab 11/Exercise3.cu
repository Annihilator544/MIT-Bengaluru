
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void inclusiveScan(int *input, int *output) {
    __shared__ int temp[N * 2];

    int tx = threadIdx.x;
    int offset = 1;

    temp[2 * tx] = input[2 * tx];
    temp[2 * tx + 1] = input[2 * tx + 1];

    for (int d = N >> 1; d > 0; d >>= 1) {
        __syncthreads();

        if (tx < d) {
            int ai = offset * (2 * tx + 1) - 1;
            int bi = offset * (2 * tx + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (tx == 0) {
        temp[N * 2 - 1] = 0; // Set the last element to 0 for exclusive scan
    }

    for (int d = 1; d < N * 2; d *= 2) {
        offset >>= 1;
        __syncthreads();

        if (tx < d) {
            int ai = offset * (2 * tx + 1) - 1;
            int bi = offset * (2 * tx + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }

    __syncthreads();

    output[2 * tx] = temp[2 * tx];
    output[2 * tx + 1] = temp[2 * tx + 1];
}

int main() {
    int input[N] = {3, 1, 7, 0, 4, 1, 6, 3, 2, 5};
    int output[N];

    int *d_input, *d_output;
    int size = N * sizeof(int);

    // Allocate memory on device
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_output, size);

    // Copy input data from host to device
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    // Launch kernel
    inclusiveScan<<<1, N/2>>>(d_input, d_output);

    // Copy result back from device to host
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    // Display result
    printf("Input Array:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", input[i]);
    }
    printf("\n\nInclusive Scan Result:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", output[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
