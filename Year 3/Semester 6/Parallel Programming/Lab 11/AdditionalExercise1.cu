
#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX_ITEMS 5
#define MAX_FRIENDS 3

__global__ void calculateTotalPurchase(float *prices, int *purchases, float *total) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    atomicAdd(total, prices[tid] * purchases[tid]);
}

int main() {
    float prices[MAX_ITEMS] = {10.5, 20.25, 15.75, 8.0, 12.5};
    int purchases[MAX_ITEMS * MAX_FRIENDS] = {2, 1, 0, 3, 1, 2, 0, 1, 2, 3, 0, 1};  // Example purchases by each friend
    float totalPurchase = 0.0;

    float *d_prices, *d_total;
    int *d_purchases;
    int itemsSize = MAX_ITEMS * sizeof(float);
    int friendsSize = MAX_ITEMS * MAX_FRIENDS * sizeof(int);

    // Allocate memory on device
    hipMalloc((void **)&d_prices, itemsSize);
    hipMalloc((void **)&d_purchases, friendsSize);
    hipMalloc((void **)&d_total, sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_prices, prices, itemsSize, hipMemcpyHostToDevice);
    hipMemcpy(d_purchases, purchases, friendsSize, hipMemcpyHostToDevice);
    hipMemset(d_total, 0, sizeof(float));  // Initialize total to 0 on device

    // Launch kernel
    calculateTotalPurchase<<<1, MAX_ITEMS>>>(d_prices, d_purchases, d_total);

    // Copy result back from device to host
    hipMemcpy(&totalPurchase, d_total, sizeof(float), hipMemcpyDeviceToHost);

    // Display result
    printf("Total Purchase by %d friends: $%.2f\n", MAX_FRIENDS, totalPurchase);

    // Free device memory
    hipFree(d_prices);
    hipFree(d_purchases);
    hipFree(d_total);

    return 0;
}
