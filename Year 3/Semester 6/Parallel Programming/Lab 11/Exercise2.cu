
#include <hip/hip_runtime.h>
#include <stdio.h>

#define WIDTH 10
#define MASK_WIDTH 3
#define TILE_WIDTH 4

__global__ void convolution(float *input, float *mask, float *output, int width, int mask_width) {
    __shared__ float tile[TILE_WIDTH + MASK_WIDTH - 1];

    int tx = threadIdx.x;
    int gx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load tile from global memory into shared memory
    if (gx >= 0 && gx < width)
        tile[tx] = input[gx];
    else
        tile[tx] = 0;

    __syncthreads();

    float result = 0;
    for (int i = 0; i < mask_width; i++) {
        result += tile[tx + i] * mask[i];
    }

    // Write the result back to global memory
    if (gx >= 0 && gx < width)
        output[gx] = result;
}

int main() {
    float input[WIDTH] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    float mask[MASK_WIDTH] = {0.5, 1, 0.5};
    float output[WIDTH];

    float *d_input, *d_mask, *d_output;
    int size = WIDTH * sizeof(float);
    int mask_size = MASK_WIDTH * sizeof(float);

    // Allocate memory on device
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_mask, mask_size);
    hipMalloc((void **)&d_output, size);

    // Copy data from host to device
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, mask_size, hipMemcpyHostToDevice);

    // Launch kernel
    convolution<<<1, WIDTH>>>(d_input, d_mask, d_output, WIDTH, MASK_WIDTH);

    // Copy result back to host
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);

    // Display result
    printf("Convolution Result:\n");
    for (int i = 0; i < WIDTH; i++) {
        printf("%.2f ", output[i]);
    }
    printf("\n");

    return 0;
}
