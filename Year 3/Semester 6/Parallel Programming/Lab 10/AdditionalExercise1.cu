
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define M 4
#define N 4
#define THREADS_PER_BLOCK 256

__global__ void computeMatrixB(int *matrixA, int *matrixB, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int sum = 0;
        // Compute sum of elements in ith row
        for (int j = 0; j < cols; j++) {
            sum += matrixA[row * cols + j];
        }
        // Compute sum of elements in jth column
        for (int i = 0; i < rows; i++) {
            sum += matrixA[i * cols + col];
        }
        // Store the sum in matrixB
        matrixB[row * cols + col] = sum;
    }
}

int main() {
    int matrixA[M][N] = {{1, 2, 3, 4},
                         {5, 6, 7, 8},
                         {9, 10, 11, 12},
                         {13, 14, 15, 16}};
    int matrixB[M][N];

    int *d_matrixA, *d_matrixB;
    int size = M * N * sizeof(int);

    // Allocate memory for matrix A and matrix B on device
    hipMalloc((void **)&d_matrixA, size);
    hipMalloc((void **)&d_matrixB, size);

    // Copy matrix A to device
    hipMemcpy(d_matrixA, matrixA, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(16, 16); // 16x16 threads per block
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);

    // Launch kernel to compute matrix B
    computeMatrixB<<<gridSize, blockSize>>>(d_matrixA, d_matrixB, M, N);

    // Copy matrix B back to host
    hipMemcpy(matrixB, d_matrixB, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_matrixA);
    hipFree(d_matrixB);

    // Display resultant matrix B
    printf("Resultant Matrix B:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", matrixB[i][j]);
        }
        printf("\n");
    }

    return 0;
}
