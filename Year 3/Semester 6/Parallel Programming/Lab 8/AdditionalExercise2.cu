
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MAX_STRING_LENGTH 1024
#define MAX_OUTPUT_LENGTH (MAX_STRING_LENGTH * 10) // Maximum output length assuming N up to 10
#define THREADS_PER_BLOCK 256

__global__ void concatenateStrings(const char *inputString, int inputLength, char *outputString, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < inputLength * N) {
        int inputIndex = tid % inputLength;
        outputString[tid] = inputString[inputIndex];
    }
}

int main() {
    const char *inputString = "Hello";
    int N = 3;

    char *d_inputString, *d_outputString;
    int inputLength = strlen(inputString);
    int outputLength = inputLength * N;
    char outputString[MAX_OUTPUT_LENGTH];

    // Allocate memory for inputString and outputString on device
    hipMalloc((void **)&d_inputString, inputLength * sizeof(char));
    hipMalloc((void **)&d_outputString, outputLength * sizeof(char));

    // Copy inputString to device
    hipMemcpy(d_inputString, inputString, inputLength * sizeof(char), hipMemcpyHostToDevice);

    int numBlocks = (outputLength + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel to concatenate strings
    concatenateStrings<<<numBlocks, THREADS_PER_BLOCK>>>(d_inputString, inputLength, d_outputString, N);

    // Copy outputString back to host
    hipMemcpy(outputString, d_outputString, outputLength * sizeof(char), hipMemcpyDeviceToHost);

    // Null-terminate the output string
    outputString[outputLength] = '\0';

    // Cleanup
    hipFree(d_inputString);
    hipFree(d_outputString);

    // Display result
    printf("Input string Sin:   %s\n", inputString);
    printf("N:                   %d\n", N);
    printf("Output string Sout:  %s\n", outputString);

    return 0;
}
