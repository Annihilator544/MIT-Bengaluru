
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MAX_STRING_LENGTH 1000

__global__ void repeatString(char *inputString, char *outputString, int stringLength) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < stringLength) {
        int count = 0;
        for(int i=0 ;i<stringLength;i++){
        for(int j=0 ;j<i+1;j++){
            outputString[count] = inputString[i];
            count++;
        }}
    }
}

int main() {
    char inputString[MAX_STRING_LENGTH];
    char outputString[MAX_STRING_LENGTH * 3]; // Adjusted for triple length
    char *d_inputString, *d_outputString;
    int stringLength;

    printf("Enter a string: ");
    fgets(inputString, MAX_STRING_LENGTH, stdin);
    // Remove newline character if present
    if (inputString[strlen(inputString) - 1] == '\n') {
        inputString[strlen(inputString) - 1] = '\0';
    }

    stringLength = strlen(inputString);

    // Allocate memory for inputString and outputString on device
    hipMalloc((void **)&d_inputString, stringLength * sizeof(char));
    hipMalloc((void **)&d_outputString, stringLength * 3 * sizeof(char)); // Triple length

    // Copy inputString to device
    hipMemcpy(d_inputString, inputString, stringLength * sizeof(char), hipMemcpyHostToDevice);

    // Calculate the number of blocks needed
    int numBlocks = (stringLength + 255) / 256;

    // Launch kernel
    repeatString<<<numBlocks, 256>>>(d_inputString, d_outputString, stringLength);

    // Copy outputString back to host
    hipMemcpy(outputString, d_outputString, stringLength * 3 * sizeof(char), hipMemcpyDeviceToHost); // Triple length

    // Cleanup
    hipFree(d_inputString);
    hipFree(d_outputString);

    // Display result
    printf("Input string S:   %s\n", inputString);
    printf("Output string RS:  %s\n", outputString);

    return 0;
}
