
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MAX_STRING_LENGTH 1000
#define MAX_WORD_LENGTH 50

__global__ void countWord(char *sentence, char *word, int *count, int sentenceLength, int wordLength) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < sentenceLength) {
        if (tid + wordLength <= sentenceLength) {
            int match = 1;
            for (int i = 0; i < wordLength; i++) {
                if (sentence[tid + i] != word[i]) {
                    match = 0;
                    break;
                }
            }
            if (match == 1) {
                atomicAdd(count, 1);
            }
        }
    }
}

int main() {
    char sentence[MAX_STRING_LENGTH];
    char word[MAX_WORD_LENGTH];
    int *d_count, *count;
    int sentenceLength, wordLength, countValue = 0;
    char *d_sentence, *d_word;

    printf("Enter a sentence: ");
    fgets(sentence, MAX_STRING_LENGTH, stdin);
    // Remove newline character if present
    if (sentence[strlen(sentence) - 1] == '\n') {
        sentence[strlen(sentence) - 1] = '\0';
    }

    printf("Enter the word to count: ");
    scanf("%s", word);

    sentenceLength = strlen(sentence);
    wordLength = strlen(word);

    // Allocate memory for count on host and device
    count = (int *)malloc(sizeof(int));
    hipMalloc((void **)&d_count, sizeof(int));
    hipMemcpy(d_count, &countValue, sizeof(int), hipMemcpyHostToDevice);

    // Allocate memory for sentence and word on device
    hipMalloc((void **)&d_sentence, sentenceLength * sizeof(char));
    hipMalloc((void **)&d_word, wordLength * sizeof(char));
    hipMemcpy(d_sentence, sentence, sentenceLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, wordLength * sizeof(char), hipMemcpyHostToDevice);

    // Calculate the number of blocks needed
    int numBlocks = (sentenceLength + 255) / 256;

    // Launch kernel
    countWord<<<numBlocks, 256>>>(d_sentence, d_word, d_count, sentenceLength, wordLength);

    // Copy count back to host
    hipMemcpy(&countValue, d_count, sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    free(count);
    hipFree(d_count);
    hipFree(d_sentence);
    hipFree(d_word);

    // Display result
    printf("Number of occurrences of '%s' in the sentence: %d\n", word, countValue);

    return 0;
}